#include "hip/hip_runtime.h"
#include <iostream>
#include "cufft_callback.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <criterion/criterion.h>
#include <criterion/logging.h>

using namespace cufftcallback;

// used with custom type
/********************************/
/* SCALE USING A CUFFT CALLBACK */
/********************************/

struct Scale
{
  enum {eParamNum = 0};
  __device__ float2 operator()(ParPos<0> l,float2& e,size_t of,void* i)
  {
    return e*0.25f;
  }
};

#define CALL_FFT() \
  checkCudaErrors(hipfftExecC2C(plan, thrust::raw_pointer_cast(d_vec.data()) \
        ,thrust::raw_pointer_cast(d_vec.data()), HIPFFT_FORWARD));

/*
 * Test: Using of a custom functor to scale
 */
// Register callback
CUFFT_STORE_CALLBACK_REG_CUSTOM(scale_025,Scale);
Test(cufft_callbacks,custom_scale)
{
  const int N=48;
  // --- Setting up input device vector
  thrust::device_vector<float2> d_vec;
  cr_assert_none_throw(d_vec.resize(N,make_hipComplex(1.0f,2.0f)));
  // --- Callback initialization
  CUFFT_STORE_CALLBACK_INIT_CUSTOM(scale_025,Scale);
  //Create cufft plan
  hipfftHandle plan;
  hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
  // --- Apply the callback
  cuFftStoreCallback::apply<Scale>(plan);
  // --- Perform in-place direct Fourier transform
  CALL_FFT();
  // --- Setting up output host vector
  thrust::host_vector<float2> h_vec;
  cr_expect_none_throw(h_vec.resize(N));
  cr_expect_none_throw(h_vec = d_vec);
  cr_expect_eq(h_vec[0], make_hipComplex(12.,24.));
  cr_expect_eq(h_vec[1], make_hipComplex(0.0f,0.0f));
  //Clean up
  checkCudaErrors(hipfftDestroy(plan));
}

/*
 * Test: multiply with an array of floats [*=f_i]
 */
// Register callback
CUFFT_STORE_CALLBACK_REG(mul_float,CFloat()*=FloatUser());
Test(cufft_callbacks,mul_float)
{
  const int N=48;
  // --- Setting up input device vector
  thrust::device_vector<float2> d_vec;
  cr_assert_none_throw(d_vec.resize(N,make_hipComplex(1.0f,2.0f)));
  // --- Setting up scaling device vector
  thrust::device_vector<float> d_vecSc;
  cr_assert_none_throw(d_vecSc.resize(N,0.5f));
  // --- Callback initialization
  CUFFT_STORE_CALLBACK_INIT(mul_float);
  //Create cufft plan
  hipfftHandle plan;
  hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
  // --- Apply the callback
  cuFftStoreCallback::apply<REG_TYPE(mul_float)>
  (plan,thrust::raw_pointer_cast(d_vecSc.data()));
  // --- Perform in-place direct Fourier transform
  CALL_FFT();
  // --- Setting up output host vector
  thrust::host_vector<float2> h_vec(N);
  h_vec = d_vec;
  cr_expect_eq(h_vec[0], make_hipComplex(24.,48.));
  cr_expect_eq(h_vec[1], make_hipComplex(0.0f,0.0f));
  //Clean up
  checkCudaErrors(hipfftDestroy(plan));
}

/*
 * Test: multiply with an array of complex floats [*=c_i]
 */
CUFFT_STORE_CALLBACK_REG(mul_cf,CFloat()*=CFloatUser());
Test(cufft_callbacks,mul_cf)
{
  const int N=48;
  // --- Setting up input device vector
  thrust::device_vector<float2> d_vec;
  cr_assert_none_throw(d_vec.resize(N,make_hipComplex(1.0f,2.0f)));
  // --- Setting up scaling device vectors
  thrust::device_vector<float2> d_vecC;
  cr_assert_none_throw(d_vecC.resize(N,make_hipComplex(0.01f,0.1f)));
  // --- Callback initialization
  CUFFT_STORE_CALLBACK_INIT(mul_cf);
  //Create cufft plan
  hipfftHandle plan;
  hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
  // --- Apply the callback
  cuFftStoreCallback::apply<REG_TYPE(mul_cf)>
  (plan,thrust::raw_pointer_cast(d_vecC.data()));
  // --- Perform in-place direct Fourier transform
  CALL_FFT();
  // --- Setting up output host vector
  thrust::host_vector<float2> h_vec(N);
  h_vec = d_vec;
  //criterion_info(" (%f,%f)\n", h_vec[0].x,h_vec[0].y);
  cr_expect_eq(h_vec[0], make_hipComplex(-9.12f,5.76f));
  cr_expect_eq(h_vec[1], make_hipComplex(0.0f,0.0f));
  //Clean up
  checkCudaErrors(hipfftDestroy(plan));
}

/*
 * Test: multiply with itself [l*=l]
 */
CUFFT_STORE_CALLBACK_REG(mul_l,CFloat()*=CFloat());
Test(cufft_callbacks,mul_l)
{
  const int N=48;
  // --- Setting up input device vector
  thrust::device_vector<float2> d_vec;
  cr_assert_none_throw(d_vec.resize(N,make_hipComplex(1.0f,2.0f)));
  // --- Callback initialization
  CUFFT_STORE_CALLBACK_INIT(mul_l);
  //Create cufft plan
  hipfftHandle plan;
  hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
  // --- Apply the callback
  cuFftStoreCallback::apply<REG_TYPE(mul_l)>
  (plan);
  // --- Perform in-place direct Fourier transform
  CALL_FFT();
  // --- Setting up output host vector
  thrust::host_vector<float2> h_vec(N);
  h_vec = d_vec;
  cr_expect_eq(h_vec[0], make_hipComplex(-6912.f,9216.f));
  cr_expect_eq(h_vec[1], make_hipComplex(0.0f,0.0f));
  //Clean up
  checkCudaErrors(hipfftDestroy(plan));
}

/*
 * Test: *=f1_i-f2_i
 */
CUFFT_STORE_CALLBACK_REG(mul_f1_MINUS_f2,CFloat()*=FloatUser()-FloatUser());
Test(cufft_callbacks,mul_float_minus_float)
{
  const int N=48;
  // --- Setting up input device vector
  thrust::device_vector<float2> d_vec;
  cr_assert_none_throw(d_vec.resize(N,make_hipComplex(1.0f,2.0f)));
  // --- Setting up scaling device vector
  thrust::device_vector<float> d_vecSc1;
  cr_assert_none_throw(d_vecSc1.resize(N,0.2f));
  thrust::device_vector<float> d_vecSc2;
  cr_assert_none_throw(d_vecSc2.resize(N,0.1f));
  // --- Callback initialization
  CUFFT_STORE_CALLBACK_INIT(mul_f1_MINUS_f2);
  //Create cufft plan
  hipfftHandle plan;
  hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
  // --- Apply the callback
  cuFftStoreCallback::apply<REG_TYPE(mul_f1_MINUS_f2)>
  (plan,thrust::raw_pointer_cast(d_vecSc1.data())
     ,thrust::raw_pointer_cast(d_vecSc2.data()));
  // --- Perform in-place direct Fourier transform
  CALL_FFT();
  // --- Setting up output host vector
  thrust::host_vector<float2> h_vec(N);
  h_vec = d_vec;
  cr_expect_eq(h_vec[0], make_hipComplex(4.8f,9.6f));
  cr_expect_eq(h_vec[1], make_hipComplex(0.0f,0.0f));
  //Clean up
  checkCudaErrors(hipfftDestroy(plan));
}

/*
 * Test: *=fc*(f1_i-f2_i)
 */
CUFFT_STORE_CALLBACK_REG(mul_f1c_mul_f2_minus_f3
    ,CFloat()*=FloatUserScalar()*(FloatUser()-FloatUser()));
Test(cufft_callbacks,mul_fc_mul_float_minus_float)
{
  const int N=48;
  // --- Setting up input device vector
  thrust::device_vector<float2> d_vec;
  cr_assert_none_throw(d_vec.resize(N,make_hipComplex(1.0f,2.0f)));
  // --- Setting up scaling device vectors
  thrust::device_vector<float> d_Sc0;
  cr_assert_none_throw(d_Sc0.resize(1,0.001f));
  thrust::device_vector<float> d_vecSc1;
  cr_assert_none_throw(d_vecSc1.resize(N,150.f));
  thrust::device_vector<float> d_vecSc2;
  cr_assert_none_throw(d_vecSc2.resize(N,25.0f));
  // --- Callback initialization
  CUFFT_STORE_CALLBACK_INIT(mul_f1c_mul_f2_minus_f3);
  //Create cufft plan
  hipfftHandle plan;
  hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
  // --- Apply the callback
  cuFftStoreCallback::apply<REG_TYPE(mul_f1c_mul_f2_minus_f3)>
  (plan,thrust::raw_pointer_cast(d_Sc0.data())
   ,thrust::raw_pointer_cast(d_vecSc1.data())
   ,thrust::raw_pointer_cast(d_vecSc2.data()));
  // --- Perform in-place direct Fourier transform
  CALL_FFT();
  // --- Setting up output host vector
  thrust::host_vector<float2> h_vec(N);
  h_vec = d_vec;
  cr_expect_eq(h_vec[0], make_hipComplex(6.0f,12.0f));
  cr_expect_eq(h_vec[1], make_hipComplex(0.0f,0.0f));
  //Clean up
  checkCudaErrors(hipfftDestroy(plan));
}
